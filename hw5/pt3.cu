#include "hip/hip_runtime.h"
// includes, system
#include <stdio.h>
#include <assert.h>

#define ARRAY_SIZE 2000000
#define STRING_SIZE 16
int char_array[ARRAY_SIZE*STRING_SIZE];
int char_counts[26];

char getRandomChar()
{
	int randNum = 0;
	char randChar = ' ';

	randNum = 26 * (rand() / (RAND_MAX + 1.0)); 	// pick number 0 < # < 25
	randNum = randNum + 97;				// scale to 'a'
	randChar = (char) randNum;

	// printf("%c", randChar);
	return randChar;
}

void init_arrays()
{
  int i, j, randNum;
  char randChar;

  for ( i = 0; i < ARRAY_SIZE *STRING_SIZE; i++) {
		 char_array[i] = (int)getRandomChar();
	
  }

  for ( i = 0; i < 26; i++ ) {
  	char_counts[i] = 0;
  }
}

void checkCUDAError(const char* msg);


__global__ void countChar(int *d_out, int *d_in, int perThread, int numThreads)
{
    int size = ARRAY_SIZE*STRING_SIZE;
    
    int i;
    int inOffset  = blockDim.x * blockIdx.x;
    int in  = inOffset + threadIdx.x;
    int theChar;
    int charLoc;
    for(i = 0; i < perThread; i++)
    {
        theChar = d_in[in*perThread+i];
        charLoc = theChar - 97;
        d_out[in*26+charLoc]++;
    }
    
}

void print_results(int* results, int totalRuns)
{
    int i, j, total = 0, count;
    for(i = 0; i < 26; i++)
    {
        count = 0;
        for(j = 0; j < totalRuns; j++)
        {
            count += results[j*26+i];            
        }
        total += count;
        printf(" %c %d\n", (char) (i + 97), count);
    }
    printf("\nTotal characters:  %d\n", total);
}

/////////////////////////////////////////////////////////////////////
// Program main
/////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{

    int totalSize = ARRAY_SIZE * STRING_SIZE; 
    int perThread = 10000;
    int size = totalSize/perThread;
    int *deviceInput, *deviceAnswer;
    int numThreadsPerBlock = 100;
    int numBlocks =  size / numThreadsPerBlock; 

    int totalThreads = numThreadsPerBlock * numBlocks; 

    // allocate host and device memory
    size_t memSize = ARRAY_SIZE*STRING_SIZE * sizeof(int);
    hipMalloc((void **) &deviceInput, memSize );
    hipMalloc((void **) &deviceAnswer, totalThreads * 26 * sizeof(int));
    int* localReturn = (int*)malloc(sizeof(int)*26*totalThreads);

    
    init_arrays();
    
    hipMemcpy( deviceInput, char_array, memSize, hipMemcpyHostToDevice);
    hipMemset( deviceAnswer, 0, totalThreads*26*sizeof(int));
    // launch kernel
    dim3 dimGrid(numBlocks);
    dim3 dimBlock(numThreadsPerBlock);
    countChar <<< dimGrid, dimBlock >>>( deviceAnswer, deviceInput, perThread, totalThreads);

    
    hipDeviceSynchronize();
    checkCUDAError("kernel invocation");

    hipMemcpy( localReturn , deviceAnswer, totalThreads*26*sizeof(int), hipMemcpyDeviceToHost );
    checkCUDAError("memcpy");

    hipFree(deviceAnswer);
    hipFree(deviceInput);

    print_results(localReturn, totalThreads);

    return 0;
}
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, 
                             hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}

